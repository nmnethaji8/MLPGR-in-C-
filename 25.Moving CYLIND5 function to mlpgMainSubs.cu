#include "hip/hip_runtime.h"
#include<iostream>
#include<fstream>
#include<string>
#include<vector>
#include<math.h>

#include"modules_v3.1.h"
#include"fnptCoupling.h"
#include"modCommon.h"
#include"nodelinkNew_v2.3.h"
#include"mlpgMainSubs.h"

using namespace std;
using namespace PROBESMOD;
using namespace FNPTCPLMOD;
using namespace COMMONMOD;
using namespace NODELINKMOD;

void NODEGRID(string MESHFILE, long int &FSNOD1, long int &FSNOD2,
double *DOMX, double *DOMY, double *DOMZ, double CYLX, double CYLY,
double CYLR, double DDL )
{
    long int I, J, NGHST, IEND;
    CYLIND5(MESHFILE, FSNOD1, FSNOD2, DOMX, DOMY, DOMZ, 
    CYLX, CYLY, CYLR, DDL);

    ofstream mlpgTerOut;
    mlpgTerOut.open("mlpgTerOut.dat", ofstream::app);
    mlpgTerOut << "TOTAL WATER PARTICLE NUMBER IS,\t" << NODEID[2] << endl;
    mlpgTerOut << "TOTAL WATERANDINNER WALL PARTICLE NUMBER,\t" << NODEID[1] << endl;
    mlpgTerOut << "TOTAL  PARTICLE NUMBER IS,\t" << NODEID[0] << endl;
    if(NODEID[0]>LNODE)
    {
        cout<<"ERROR IN ALLOCATION OF NUMBER OF NODES"<<endl;
    }

    // Finding unique boundary nodes
    for(long int i=NODEID[2];i<NODEID[0];i++)
    {
        for(long int j=i+1;j<NODEID[0];j++)
        {
            if((COORX[i]==COORX[j])&&(COORY[i]==COORY[j])&&(COORZ[i]==COORZ[j]))
            {
                cout << "unique node found\t" << i+1 << " " << j+1 << " "
                << NODEID2[i] << " " << NODEID2[j] << " " << endl;

                cout << COORX[i] << " " << COORY[i] << " "
                << COORZ[i] << endl;

                cout << COORX[j] << " " << COORY[j] << " "
                << COORZ[j] << endl;
            }
        }
    }

    ICELLX = (long int)(IXMAX/RCELL)+1;
    ICELLY = (long int)(IYMAX/RCELL)+1;
    ICELLZ = (long int)(IZMAX*2/RCELL)+1;

    mlpgTerOut << "CELL SIZES" << " " << ICELLX << " " 
    << ICELLY << " " <<  ICELLZ << endl;
    mlpgTerOut.close();

    NGHST=0;
    ofstream OutputXY_FLUENT;
    OutputXY_FLUENT.open("Output/XY_FLUENT.DAT"); //
    for(long int i=NODEID[1];i<NODEID[0];i++)
    {
        if(NODEID2[i]==-9)
        {
            NGHST++;
        }
    }
    OutputXY_FLUENT << NODEID[1]+NGHST << endl;
    OutputXY_FLUENT << "TIME=" << 0 << endl;
    for(long int i=0;i<NODEID[1];i++)
    {
        if((fabs(COORX[i])<=70)&&(fabs(COORY[2*LNODE+i])<=70)&&(fabs(COORZ[2*LNODE+i])<=70))
        {
            OutputXY_FLUENT << COORX[i] << " " << COORY[i] << " "
            << COORZ[i] << " " << " 0 " << " " << SNX[i] << " " << SNY[i] << " "
            << SNZ[i] << " " << NODEID2[i] << endl;
        }
        else
        {
            OutputXY_FLUENT << "-10 -10 -10 0 0 0 0 " << NODEID2[i] << endl;
        }   
    }

    for(long int i=NODEID[1];i<NODEID[0];i++)
    {
        if(NODEID2[i]==-9)
        {
            OutputXY_FLUENT << COORX[i] << " " << COORY[i] << " "
            << COORZ[i] << " 0 0 0 0 " << NODEID2[i] << endl;
        }
    }
    OutputXY_FLUENT.close();
}
int main()
{
    //declaring and initializing variables from mlpgrInput.dat file
    double H0=0.7;                  //Water depth
    double DDL=0.04375;             //Avg.distance between the nodes
    double SCALE=1.55;              //Scale factor to determine the domain of influence 
    long int KW=1;                  //Coefficient for Gauss Weight Function
    long int MBAS=4;                //Number of components in base function
    double DT=0.0075, TOTAL_TIME=0; //Time-Step (s), Starting time (s)
    long int NSTEPS=5500;           //Number of time-steps

    /*READ RECORD DATA TIME,THE FREQUENCY OF PRINT OUT IN THE NUMBER OF TYPE STEPS 
    RECORD THE TIME STEP, BEFORE THAT TIME, THE RECORD FREQUENCY IS EVRSTEP,AFTER
    THAT TIME, THE RECORD FREQUENCY IS EVRSTEP1*/

    long int IPRINT=5500, I_PF=120, I_PF1=40;//Freq1, Freq1, Freq2
    long int RESFREQ=800;           //Resume file interval (number of time-step)
    bool RESUMECHK=0;
    string RESUMEFILE="Output/Resume_000000800.dat";
    long int I_CAL_V=0;             //Viscosity On? (0/1)
    double VCOEFF=0.000001;         //Kinematic viscosity
    long int I_WM=15;               //Wave-maker type (0-No wavemaker, 1-Flap, 2-piston, 15-FNPT)
    long int IFSI=0;                //Enable elastic structure? (0/1)
    long int NTHR=8;                //Number of OpenMP Threads
    long int II=170000;       //Maximum number of nodes (LNODE)
    string MESHFILE="EmptyTank_L21_dr043_botFul3.dat";//Name of mesh-file
    double DOMX[2]={0,21},DOMY[2]={0,0.7},DOMZ[2]={0,0.7};//Domain coordinates
    double CYLX=6.5,CYLY=0.7,CYLR=0.04375;//Cylinder coordinates
    double SPONGEX=13;              //X Location of Sponge layer start (before right-wall)
    long int REMESHFREQ=30;         //Remeshing Freq (=0 to disable)
    long int NP=9;

    //checking the input data
    ofstream mlpgTerOut;
    mlpgTerOut.open("mlpgTerOut.dat");
    mlpgTerOut<<"3D WATER WAVE PROBLEM WITH MLPGR METHOD\n"
    <<H0<<"\n"<<DDL<<"\n"<<SCALE<<"\n"<<KW<<"\n"<<MBAS<<"\n"
    <<DT<<"\t"<<TOTAL_TIME<<"\t"<<NSTEPS<<"\n"<<IPRINT<<"\t"<<I_PF
    <<"\t"<<I_PF1<<"\n"<<I_CAL_V<<"\t"<<VCOEFF<<"\n"<<I_WM<<"\n"
    <<IFSI<<"\n"<<NTHR<<"\n"<<MESHFILE<<"\n"<<DOMX[0]<<"\t"<<DOMY[0]<<"\t"
    <<DOMZ[0]<<"\n"<<DOMX[1]<<"\t"<<DOMY[1]<<"\t"<<DOMZ[1]<<"\n"
    <<CYLX<<"\t"<<CYLY<<"\t"<<CYLR<<"\n"<<SPONGEX<<endl;
    mlpgTerOut.close();

    //<<REMESHFREQ<<"\n"<<"\n"<<RESFREQ<<"\t"<<RESUMECHK<<"\n"<<RESUMEFILE<<II<<"\n"
    FNPTCPLTYP FP;
    NODELINKTYP MLDOM, FSDOM, BOTOM;
    PROBETYP WP,PP;

    WP.INITPROBE(NP, 1, 602);       //Number of wave probes
    //WP.FILE=602;
    ifstream mlpgrInput;
    mlpgrInput.open("mlpgrInput.dat");
    /*for(long int i=0; i<NP; i++)
    {
        for(long int j=0; j<3; j++)
        {
            cout << WP.XYZ[i*3+j] << " " ;
        }
        cout << "\n" << endl;
    }*/
    for(long int i=0; i<NP; i++)
    {
        for(long int j=0; j<2; j++)
        {
            mlpgrInput >> WP.XYZ[i*2+j];
            //cout << WP.XYZ[i*2+j] << " " ;
        }
        //cout << "\n" << endl;
    }
    //cout << WP.FILE << endl;
    mlpgrInput >> NP;
    PP.INITPROBE(NP, 1, 603);
    for(long int i=0; i<NP; i++)
    {
        for(long int j=0; j<3; j++)
        {
            mlpgrInput >>PP.XYZ[i*3+j];
        }
    }
    mlpgrInput.close();
    FP.FILENAME="/Output PHIT2 23001.dt0075.S18.dat";
    //cout<<FP.FILENAME<<endl;
    FP.NX=121;
    //cout<<FP.NX<<endl;
    FP.NY=21;
    //cout<<FP.NY<<endl;
    FP.DDL=0.05;
    //cout<<FP.DDL<<endl;
    FP.X0=18.38;
    //cout<<FP.X0<<endl;
    FP.RLXLEN=1;
    //cout<<FP.RLXLEN<<endl;
    FP.FILE=601;
    //cout<<FP.FILE<<endl;
    FP.NN=(FP.NX)*(FP.NY);
    //cout<<FP.NN<<endl;
     LNODE=II;
     INITCOMMONMOD();

    //Variables in malpgrMain

    double *FB = NULL, *PTMP = NULL, *P = NULL;
    FB = new double[ LNODE]();
    PTMP = new double[ LNODE]();
    P = new double[ LNODE]();
    double *UN = NULL, *UM = NULL;
    UN = new double[ LNODE*3]();
    UM = new double[ LNODE]();
    double *CSUXT1 = NULL, *CSUYT1 = NULL, *CSUZT1 = NULL;
    CSUXT1 = new double[ LNODE]();
    CSUYT1 = new double[ LNODE]();
    CSUZT1 = new double[ LNODE]();
    double *ERRTMP = NULL, *MMTOIM = NULL;
    ERRTMP = new double[LNODE], MMTOIM = new double[LNODE];
    long int FSNOD1 = 0, FSNOD2 = 0;
    long int NODN = 0;
    ofstream output;
    output.open("output2.txt");
    mlpgTerOut.open("mlpgTerOut.dat",ofstream::app);

    //Generate Nodes

    NODEGRID(MESHFILE, FSNOD1, FSNOD2, DOMX, DOMY, DOMZ, CYLX, CYLY, CYLR, DDL);
    for(long int i=0;i<LNODE;i++)
    {
        COORX[1*LNODE+i]=COORX[i];
        COORY[1*LNODE+i]=COORY[i];
        COORZ[1*LNODE+i]=COORZ[i];
    }

    NODN = NODEID[0];   //Total Number of Nodes
    cout << NODEID[0];

    mlpgTerOut << "[INF] FREE SURFACE REFERENCE NODES\n"
    << FSNOD1 << " " << COORX[FSNOD1-1] << " " << COORY[FSNOD1-1] << " "
    << COORZ[FSNOD1-1] << endl;
    mlpgTerOut << FSNOD2 << " " << COORX[FSNOD2-1] << " " << COORY[FSNOD2-1] << " "
    << COORZ[FSNOD2-1] << endl;

    /*MLDOM.INITCELL(ICELLX, ICELLY, ICELLZ, 500);
    FSDOM.INITCELL(ICELLX, ICELLY, 1, 100);
    BOTOM.INITCELL(ICELLX, ICELLY, 1, 100);*/

    //STORING BNDNODES FOR FORCED ADJUSTMENT IN NEWCOOR


    mlpgTerOut.close();
    return 0;
}
